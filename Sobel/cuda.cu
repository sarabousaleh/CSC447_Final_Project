#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define SOBEL_KERNEL_SIZE 3
#define TILE_SIZE 32  

__constant__ int sobel_kernel_x[SOBEL_KERNEL_SIZE][SOBEL_KERNEL_SIZE] = {{-1, 0, 1},
                                                                          {-2, 0, 2},
                                                                          {-1, 0, 1}};

__constant__ int sobel_kernel_y[SOBEL_KERNEL_SIZE][SOBEL_KERNEL_SIZE] = {{-1, -2, -1},
                                                                          { 0,  0,  0},
                                                                          { 1,  2,  1}};

__global__ void sobel_filter(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    __shared__ unsigned char shared_image[TILE_SIZE + 2][TILE_SIZE + 2];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * TILE_SIZE + tx;
    int y = blockIdx.y * TILE_SIZE + ty;

    
    if (x < width && y < height) {
        shared_image[ty + 1][tx + 1] = input_image[y * width + x];
    }

    
    if (tx == 0 && x > 0) {
        shared_image[ty + 1][0] = input_image[y * width + x - 1];
    }
    if (tx == TILE_SIZE - 1 && x < width - 1) {
        shared_image[ty + 1][TILE_SIZE + 1] = input_image[y * width + x + 1];
    }
    if (ty == 0 && y > 0) {
        shared_image[0][tx + 1] = input_image[(y - 1) * width + x];
    }
    if (ty == TILE_SIZE - 1 && y < height - 1) {
        shared_image[TILE_SIZE + 1][tx + 1] = input_image[(y + 1) * width + x];
    }

    __syncthreads();

    if (x < width && y < height) {
        int gradient_x = 0;
        int gradient_y = 0;

        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                gradient_x += shared_image[ty + i + 1][tx + j + 1] * sobel_kernel_x[i + 1][j + 1];
                gradient_y += shared_image[ty + i + 1][tx + j + 1] * sobel_kernel_y[i + 1][j + 1];
            }
        }

        output_image[y * width + x] = sqrtf((float)(gradient_x * gradient_x + gradient_y * gradient_y));
    }
}

int main() {
    int width, height, channels;
    unsigned char* input_image = stbi_load("input_image.jpg", &width, &height, &channels, 0);

    if (!input_image) {
        printf("Error loading the image!\n");
        return 1;
    }

    printf("Image loaded successfully: width=%d, height=%d, channels=%d\n", width, height, channels);

    unsigned char* d_input_image;
    unsigned char* d_output_image;

    size_t image_size = width * height;
    size_t image_bytes = image_size * sizeof(unsigned char);

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&d_input_image, image_bytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    cudaStatus = hipMalloc((void**)&d_output_image, image_bytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_input_image);
        return 1;
    }

    cudaStatus = hipMemcpy(d_input_image, input_image, image_bytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_input_image);
        hipFree(d_output_image);
        return 1;
    }

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((width + TILE_SIZE - 1) / TILE_SIZE, (height + TILE_SIZE - 1) / TILE_SIZE);

    printf("Launching CUDA kernel with gridDim=(%d, %d) blockDim=(%d, %d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

    hipEvent_t start, stop;
    float milliseconds = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    sobel_filter<<<gridDim, blockDim>>>(d_input_image, d_output_image, width, height);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Time taken by CUDA code: %.2f milliseconds\n", milliseconds);

    unsigned char* output_image = (unsigned char*)malloc(image_bytes);

    cudaStatus = hipMemcpy(output_image, d_output_image, image_bytes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_input_image);
        hipFree(d_output_image);
        free(output_image);
        return 1;
    }

    stbi_write_jpg("output_image_cuda.jpg", width, height, 1, output_image, 100);

    printf("Output image saved as output_image_cuda.jpg\n");

    stbi_image_free(input_image);
    free(output_image);

    hipFree(d_input_image);
    hipFree(d_output_image);

    return 0;
}
